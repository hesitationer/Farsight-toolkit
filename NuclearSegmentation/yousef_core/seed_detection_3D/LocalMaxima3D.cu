
#include <hip/hip_runtime.h>
__global__ void LocalMaximaKernel (float* im_vals, unsigned short* out1, int r, int c, int z, double scale_xy, double scale_z)
{
	int iGID = blockIdx.x + threadIdx.x; //global index
		
	//calculate r, c, z indices as i, j, k from global index
	int rem = ((long)iGID) % (r*c);
	int k = ((int)iGID-rem) / (r*c); 
	int j = ((long)rem) % c;
	int i = (rem-j)/c;
	
	//calculate bounds
	int min_r = (int) max(0.0,i-scale_xy);
	int min_c = (int) max(0.0,j-scale_xy);
	int min_z = (int) max(0.0,k-scale_z);
	int max_r = (int)min((double)r-1,i+scale_xy);
	int max_c = (int)min((double)c-1,j+scale_xy);                         
	int max_z = (int)min((double)z-1,k+scale_z);                         
	
	//get the intensity maximum of the bounded im_vals
	float mx = im_vals[(min_z*r*c)+(min_r*c)+min_c];
    
	for(int i = min_r; i <= max_r; i++)
    {
        for(int j = min_c; j <= max_c; j++)
        {
			for(int k = min_z; k <= max_z; k++)
			{				
				if(im_vals[(k*r*c)+(i*c)+j] > mx)
					mx = im_vals[(k*r*c)+(i*c)+j];
			}
        }
    }
	
	//if the current pixel is at the maximum intensity, set it to 255 in out1 (seedImagePtr), else set it to 0
	if(im_vals[iGID] == mx)    
		out1[iGID]=255;
	else
		out1[iGID]=0;
}

extern "C"
void Detect_Local_MaximaPoints_3D_CUDA(float* im_vals, int r, int c, int z, double scale_xy, double scale_z, unsigned short* out1)
{
}